#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define GRID_SIZE 128
#define BLOCK_SIZE 64
#define MATRIX_SIZE (BLOCK_SIZE * GRID_SIZE)

dim3 Db(BLOCK_SIZE, 1, 1);
dim3 Dg(GRID_SIZE, GRID_SIZE, 1);

hipEvent_t start, end;
float timer;

// CUDAカーネル: 2つの行列の要素ごとの掛け算
__global__ void matrix_elementwise_multiply(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    c[i * MATRIX_SIZE + j] = a[i * MATRIX_SIZE + j] * b[i * MATRIX_SIZE + j];
}

int main(int argc, char **argv) {
    hipSetDevice(0);

    int *a, *b, *c;  // 行列 a, b, c
    int *ad, *bd, *cd;  // デバイス用行列ポインタ
    int size = MATRIX_SIZE * MATRIX_SIZE * sizeof(int); // 行列のサイズ

    // イベントの作成
    hipEventCreate(&start);
    hipEventCreate(&end);

    printf("Matrix Element-wise Multiplication\n");
    printf("\nCalculation Start\n");

    // ホストメモリ上の行列の動的確保
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // 初期化
    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; ++i) {
        a[i] = b[i] = 1;
        c[i] = 0;
    }

    // デバイスメモリ上の行列の動的確保
    hipMalloc(&ad, size);
    hipMalloc(&bd, size);
    hipMalloc(&cd, size);

    // ホストからデバイスへの転送
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
    hipMemcpy(cd, c, size, hipMemcpyHostToDevice);

    // カーネル実行時間計測開始
    hipEventRecord(start, 0);

    // カーネル呼び出し
    matrix_elementwise_multiply<<<Dg, Db>>>(ad, bd, cd);

    // 結果をデバイスからホストに転送
    hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

    // カーネル実行時間計測終了
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);

    // 結果の表示
    printf("c[0][0] = %d\n", c[0]);
    printf("c[%d][%d] = %d\n", MATRIX_SIZE - 1, MATRIX_SIZE - 1, c[MATRIX_SIZE * MATRIX_SIZE - 1]);

    // 実行時間の表示
    printf("\nCalculation End\n");
    printf("\nProcessing Time : %.3f [msec]\n", timer);

    // メモリの解放
    free(a);
    free(b);
    free(c);
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    
    // イベントの破棄
    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
