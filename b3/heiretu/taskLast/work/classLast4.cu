#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>


#define grid 64
#define block 32

#define Nx block*grid
#define Ny block*grid


#define Db_x block
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);

hipEvent_t start, end;
float timer;

// __global__ void adder(int *vecd) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	int j = blockIdx.y * blockDim.y + threadIdx.y;
//     vecd[i*grid*block+j] = (i-j)*(i-j);
// }
__global__ void mult_matrix(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    for (int k=0; k<grid*block; ++k) {
        c[i*grid*block + j] += a[grid*block*i+k]*b[grid*block*k+j];
        // c[0] += 1;
    }
}


int main(int argc, char **argv) {
    hipSetDevice(0);

    // int *vec, *vecd;
    int *a, *b, *c;
    int *ad, *bd, *cd;
    int n = grid * block;  // データの数
    int size = n * n * sizeof(int); // データのサイズ

    hipEventCreate(&start);
    hipEventCreate(&end);

    printf("matrix multi\n");
    printf("\nCalculation Start\n");

    // vec = (int *)malloc(size);  // ホストメモリの確保
    a = (int *) malloc(size);
    b = (int *) malloc(size);
    c = (int *) malloc(size);

    for (int i=0; i<n*n; ++i) {
        a[i] = b[i] = 1;
        c[i] = 0;
    }
    

    // cudaMalloc(&vecd, size);  // デバイスメモリの確保
	// cudaMemcpy(vecd, vec, size, cudaMemcpyHostToDevice);
    // cudaEventRecord(start, 0);

    hipMalloc(&ad, size);
    hipMalloc(&bd, size);
    hipMalloc(&cd, size);
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
    hipMemcpy(cd, c, size, hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    

    mult_matrix<<<Dg, Db>>>(ad, bd, cd);

    // cudaMemcpy(vec, vecd, size, cudaMemcpyDeviceToHost);  // 結果のデバイスからホストへのコピー
    hipMemcpy(a, ad, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, bd, size, hipMemcpyDeviceToHost);
    hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

    // long long sum = 0;
    // for (int i = 0; i < n*n; ++i) {
    //     sum += vec[i];  // 結果の計算
    // }
    // printf("sum = %lld\n", sum);
    printf("c[%d][%d] = %d\n", 0, 0, c[0]);
    printf("c[%d][%d] = %d\n", n-1, n-1, c[n*n-1]);
    

    

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);
    printf("\nCalculation End\n");
    printf("\nProcessing Time : %.3f [msec]\n", timer);

    hipEventDestroy(start);
    hipEventDestroy(end);

    // free(vec);
    // cudaFree(vecd);

    free(a);
    free(b);
    free(c);
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    

    return 0;
}
