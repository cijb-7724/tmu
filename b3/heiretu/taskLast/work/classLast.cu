#include "hip/hip_runtime.h"
/*
 * main.cu(convert_color.cu) : �F�ϊ�
 * @ KLO lab. in TMU (2013/12/20[Fri])
 */
 

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

long long L = pow(2, 7);
#define grid 1024
#define block grid/L

#define grid 3
#define block 5

#define Nx 16
#define Ny 8

#define Db_x 8
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);


hipEvent_t start, end;	     /*  ���Ԍv���p  */
float timer;	     /*  ���Ԍv���p  */

int element = sizeof(float) * Nx * Ny;

float alpha = 2.f;
long long sum = 0;


/*
 * convert_color.cu : �F�ϊ�
 */

__global__ void convert_color (float *d_IMG_input, float *d_IMG_output)
{
	int X, Y, ID;
	X = threadIdx.x + (blockIdx.x * blockDim.x);
	Y = threadIdx.y + (blockIdx.y * blockDim.y);
	ID = X + Y * Nx;
}

__global__ void adder() {
	// sum += (i-j)*(i-j);
	sum += (blockIdx.x - threadIdx.x)*(blockIdx.x - threadIdx.x);
}
__global__ void vecAdd(double *Ad, double *Bd, double *Cd) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    Cd[i] = Ad[i] + Bd[i];
}



int main
(
	int    argc, // Argument Count
	char **argv  // Argument Vector
)
{
	// ��u�҂��Ƃ�GPU������U���Ă���̂ŁC�l��ύX���ĉ�����(0�`3)
	int GPU_Num = 2;
	hipSetDevice(GPU_Num);

	printf("Nx = %d, Ny = %d\n", Nx, Ny);
	printf("Number %d GPU working\n", GPU_Num);

	// int i, j, ID;
	
	// ���Z���Ԃ��v�����邽�߂�hipEventCreate�����s
	hipEventCreate(&start);
	hipEventCreate(&end);

	printf("\nCalculation Start\n");



	// ���Z���Ԃ��v�����邽�߂�hipEventRecord�����s���v�Z�̊J�n���L�^
	hipEventRecord(start, 0);

	//here
	adder <<< 128, 128 >>> ();
	printf("sum = %lld\n", sum);







	hipDeviceSynchronize();

    // ���Z���Ԃ��v�����邽�߂�hipEventRecord�����s���v�Z�̏I�����L�^
    hipEventRecord(end, 0);

    // ���Z���Ԃ��Z�o
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);

    printf("\nCalculation End\n");

    printf("\nProcessing Time : %.3f [msec]\n", timer);

    
    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
