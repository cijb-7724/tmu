#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>


#define grid 128
#define block 64

#define Nx block*grid
#define Ny block*grid


#define Db_x block
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);

hipEvent_t start, end;
float timer;

__global__ void adder(long *vecd) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    vecd[i*grid*block+j] = (i-j)*(i-j);
}

int main(int argc, char **argv) {
    hipSetDevice(0);

    long *vec, *vecd;
    int n = grid * block;  // データの数
    int size = n * n * sizeof(long); // データのサイズ

    hipEventCreate(&start);
    hipEventCreate(&end);

    printf("\nCalculation Start\n");

    vec = (long *)malloc(size);  // ホストメモリの確保

    hipMalloc(&vecd, size);  // デバイスメモリの確保
	hipMemcpy(vecd, vec, size, hipMemcpyHostToDevice);
    hipEventRecord(start, 0);


    adder<<<Dg ,Db>>>(vecd);  // カーネルの実行

    hipMemcpy(vec, vecd, size, hipMemcpyDeviceToHost);  // 結果のデバイスからホストへのコピー

    long long sum = 0;
    for (int i = 0; i < n*n; ++i) {
        sum += vec[i];  // 結果の計算
    }
    printf("L   = %d\n", grid*block);
    printf("sum = %lld\n", sum);

    

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);
    printf("\nCalculation End\n");
    printf("\nProcessing Time : %.3f [msec]\n", timer);

    hipEventDestroy(start);
    hipEventDestroy(end);

    free(vec);
    hipFree(vecd);

    return 0;
}
