#include "hip/hip_runtime.h"
/*
 * main.cu(convert_color.cu) : メイン
 * @ KLO lab. in TMU (2013/12/20[Fri])
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define HTH hipMemcpyHostToHost
#define HTD hipMemcpyHostToDevice
#define DTH hipMemcpyDeviceToHost
#define DTD hipMemcpyDeviceToDevice

#define Nx 1024
#define Ny 1024

#define Db_x 256
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);

float *h_IMG_input;      /* 入力画像のポインタ(Host) */
float *h_IMG_output;     /* 出力画像のポインタ(Host) */

float *d_IMG_input;      /* 入力画像のポインタ(Device) */
float *d_IMG_output;     /* 出力画像のポインタ(Device) */

hipEvent_t start, end;     /* 測定用イベント */
float timer;     /* 測定用イベント */

int element = sizeof(float) * Nx * Ny;

float alpha = 2.f;


/*
 * convert_color.cu : カーネル
 */

__global__ void convert_rotate
(
    float *d_IMG_input,
    float *d_IMG_output
)
{
    int X, Y, ID, toID;

    X = threadIdx.x + (blockIdx.x * blockDim.x);
    Y = threadIdx.y + (blockIdx.y * blockDim.y);
    ID = X + Y * Nx;
    toID = Y + (Nx - X - 1) * Nx;
    if (toID < 0 || toID >= Nx * Nx) printf("toID %d\n", toID);
    d_IMG_output[ID] = d_IMG_input[toID];
}

/*
 * メイン関数main : メイン
 */

int main
(
    int    argc, // Argument Count
    char **argv  // Argument Vector
)
{
    // 使用するGPUの選択
    int GPU_Num = 0;

    hipSetDevice(GPU_Num);

    printf("convert_rotate.cu\n");
    printf("Nx = %d, Ny = %d\n", Nx, Ny);
    printf("Number %d GPU working\n", GPU_Num);

    int i, j, ID;

    // デバイスメモリの確保
    hipMalloc((void **)&d_IMG_input, element);
    hipMalloc((void **)&d_IMG_output, element);

    hipMemset(d_IMG_input, 0, element);
    hipMemset(d_IMG_output, 0, element);

    // ホストメモリの確保
    hipHostAlloc((void **)&h_IMG_input, element, hipHostMallocPortable);
    hipHostAlloc((void **)&h_IMG_output, element, hipHostMallocPortable);

    memset(h_IMG_input, 0, element);
    memset(h_IMG_output, 0, element);

    // ファイル読み込み
    FILE *fp;     /* ファイルポインタ */

    fp = fopen("./man1024.img", "r");     /* 入力ファイルを読み込む */
    if (fp == NULL) {
        printf("ファイルを開けませんでした\n");
        return 0;
    }

    for (j = 0; j < Ny; j++) {
        for (i = 0; i < Nx; i++) {
            ID = i + j * Nx;
            fscanf(fp, "%f", &(h_IMG_input[ID]));     /* 1要素読み込んで h_IMG_input[ID] に格納 */
        }
    }

    fclose(fp);     /* ファイルをクローズ */

    // 作業開始のイベントを作成
    hipEventCreate(&start);
    hipEventCreate(&end);

    printf("\nCalculation Start\n");

    hipMemcpy(d_IMG_input, h_IMG_input, element, HTD);

    // 処理の開始時間を記録するイベントを記録
    hipEventRecord(start, 0);

    /* 画像処理 */
    convert_rotate <<<Dg, Db>>> (d_IMG_input, d_IMG_output);

    hipDeviceSynchronize();

    // 処理の終了時間を記録するイベントを記録
    hipEventRecord(end, 0);

    // イベントの同期
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);

    printf("\nCalculation End\n");

    printf("\nProcessing Time : %.3f [msec]\n", timer);

    /* ファイル出力 */
    hipMemcpy(h_IMG_output, d_IMG_output, element, DTH);

    // 出力ファイルを開く
    fp = fopen("output1024convert_gpu_5a.img", "w");
   
    if (fp == NULL) {
        printf("ファイルを開けませんでした");
        return 0;
    }

    for (j = 0; j < Ny; j++) {
        for (i = 0; i < Nx; i++) {
            ID = i + j * Nx;
            fprintf(fp, "%d\n", (unsigned char) h_IMG_output[ID]);     /* 1要素書き出す */
        }
    }

    fclose(fp);

    // デバイスメモリの解放
    hipFree(d_IMG_input);
    hipFree(d_IMG_output);

    // ホストメモリの解放
    hipHostFree(h_IMG_input);
    hipHostFree(h_IMG_output);

    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
