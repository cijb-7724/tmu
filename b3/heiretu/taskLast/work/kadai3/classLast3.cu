#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>


#define grid 128
#define block 64

#define Nx block*grid
#define Ny block*grid

#define Db_x block
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);

hipEvent_t start, end;
float timer;

__global__ void adder(long *vecd) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    vecd[i*Nx+j] = (i-j)*(i-j);
}

int main(int argc, char **argv) {
    hipSetDevice(0);

    long *vec, *vecd;
    int n = grid * block;  // �f�[�^�̐�
    int size = n * n * sizeof(long); // �f�[�^�̃T�C�Y

    hipEventCreate(&start);
    hipEventCreate(&end);

    printf("\nCalculation Start\n");

    vec = (long *)malloc(size);  // �z�X�g�������̊m��

    hipMalloc(&vecd, size);  // �f�o�C�X�������̊m��
	hipMemcpy(vecd, vec, size, hipMemcpyHostToDevice);
    hipEventRecord(start, 0);


    adder<<<Dg ,Db>>>(vecd);  // �J�[�l���̎��s

    hipMemcpy(vec, vecd, size, hipMemcpyDeviceToHost);  // ���ʂ̃f�o�C�X����z�X�g�ւ̃R�s�[

    long long sum = 0;
    for (int i = 0; i < n*n; ++i) {
        sum += vec[i];  // ���ʂ̌v�Z
    }
    printf("L   = %d\n", grid*block);
    printf("sum = %lld\n", sum);

    

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);
    printf("\nCalculation End\n");
    printf("\nProcessing Time : %.3f [msec]\n", timer);

    hipEventDestroy(start);
    hipEventDestroy(end);

    free(vec);
    hipFree(vecd);

    return 0;
}
