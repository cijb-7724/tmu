#include "hip/hip_runtime.h"
/*
 * main.cu(convert_color.cu) : 色変換
 * @ KLO lab. in TMU (2013/12/20[Fri])
 */
 

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define HTH hipMemcpyHostToHost
#define HTD hipMemcpyHostToDevice
#define DTH hipMemcpyDeviceToHost
#define DTD hipMemcpyDeviceToDevice

#define Nx 1024
#define Ny 1024

#define Db_x 256
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);

float *h_IMG_input;	     /*  入力データ用(Host)  */
float *h_IMG_output;     /*  出力データ用(Host)  */

float *d_IMG_input;	     /*  入力データ用(Device)  */
float *d_IMG_output;     /*  出力データ用(Device)  */

hipEvent_t start, end;	     /*  時間計測用  */
float timer;	     /*  時間計測用  */

int element = sizeof(float) * Nx * Ny;

float alpha = 2.f;


/*
 * convert_color.cu : 色変換
 */

__global__ void convert_color
(
	float *d_IMG_input,
	float *d_IMG_output
)
{
	int X, Y, ID;

	X = threadIdx.x + (blockIdx.x * blockDim.x);
	Y = threadIdx.y + (blockIdx.y * blockDim.y);
	ID = X + Y * Nx;
	d_IMG_output[ID] = d_IMG_input[ID];
}

/*
 * ここからmain : 色変換
 */

int main
(
	int    argc, // Argument Count
	char **argv  // Argument Vector
)
{
	// 受講者ごとにGPUが割り振られているので，値を変更して下さい(0～3)
	int GPU_Num = 0;

	hipSetDevice(GPU_Num);

	printf("convert_color.cu\n");
	printf("Nx = %d, Ny = %d\n", Nx, Ny);
	printf("Number %d GPU working\n", GPU_Num);

	int i, j, ID;

	// デバイス（VRAM内）に要素を確保
	hipMalloc((void **)&d_IMG_input, element);
	hipMalloc((void **)&d_IMG_output, element);

	hipMemset(d_IMG_input, 0, element);
	hipMemset(d_IMG_output, 0, element);

	// ホスト（RAM内）に要素を確保
	hipHostAlloc((void **)&h_IMG_input, element, hipHostMallocPortable);
	hipHostAlloc((void **)&h_IMG_output, element, hipHostMallocPortable);

	memset(h_IMG_input, 0, element);
	memset(h_IMG_output, 0, element);

//ファイルよりデータ入力
	FILE *fp;	     /*  入出ファイル用  */

	fp = fopen("./man1024.img", "r");     /*  読込みモードでファイルをオープンする  */
	if(fp == NULL) {
		printf("ファイルを開くことが出来ませんでした．\n");
		return 0;
	}
 
	for(j = 0; j < Ny; j++){
		for(i = 0; i < Nx; i++){
			ID = i + j * Nx;
			fscanf(fp, "%f", &(h_IMG_input[ID]) );     /*  1行読む　→ h_IMG_input[ID])に入れる */
		}
	}
	
	fclose(fp);     /*  ファイルをクローズする  */

	
	// 演算時間を計測するためのhipEventCreateを実行
	hipEventCreate(&start);
	hipEventCreate(&end);

	printf("\nCalculation Start\n");

	hipMemcpy(d_IMG_input, h_IMG_input, element, HTD);

	// 演算時間を計測するためのhipEventRecordを実行し計算の開始を記録
	hipEventRecord(start, 0);


	/* image processing */
	convert_color <<< Dg, Db >>> (d_IMG_input, d_IMG_output);

	hipDeviceSynchronize();

    // 演算時間を計測するためのhipEventRecordを実行し計算の終了を記録
    hipEventRecord(end, 0);

    // 演算時間を算出
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);

    printf("\nCalculation End\n");

    printf("\nProcessing Time : %.3f [msec]\n", timer);

    /* File Output */
    hipMemcpy(h_IMG_output, d_IMG_output, element, DTH);

//出力用データをファイルに書き込む
    fp = fopen("output1024convert_gpu.img", "w");     /*  書込みモードでファイルをオープンする  */
   
    if(fp == NULL){
        printf("ファイルを作れませんでした");
        return 0;  /*  ここでプログラム終了  */
    }

    for(j = 0; j < Ny; j++){
		for(i = 0; i < Nx; i++){
			ID = i + j * Nx;
			fprintf(fp, "%d\n", (unsigned char) h_IMG_output[ID]);     /*  1行書込み　→ h_IMG_output[ID])に入れる */
		}
	}

	fclose(fp);

    hipFree(d_IMG_input);
    hipFree(d_IMG_output);

    hipHostFree(h_IMG_input);
    hipHostFree(h_IMG_output);

    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
