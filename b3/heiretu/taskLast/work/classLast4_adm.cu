#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>


#define grid 16
#define block 8

#define Nx block*grid
#define Ny block*grid


#define Db_x block
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);

hipEvent_t start, end;
float timer;

// __global__ void adder(int *vecd) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	int j = blockIdx.y * blockDim.y + threadIdx.y;
//     vecd[i*grid*block+j] = (i-j)*(i-j);
// }
__global__ void adm_mult_matrix(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
    c[i*grid*block + j] += a[grid*block*i+j]*b[grid*block*i+j];
}


int main(int argc, char **argv) {
    hipSetDevice(0);

    // int *vec, *vecd;
    int *a, *b, *c;
    int *ad, *bd, *cd;
    int n = grid * block;  // �f�[�^�̐�
    int size = n * n * sizeof(int); // �f�[�^�̃T�C�Y

    hipEventCreate(&start);
    hipEventCreate(&end);

    printf("matrix adm multi\n");
    printf("\nCalculation Start\n");

    // vec = (int *)malloc(size);  // �z�X�g�������̊m��
    a = (int *) malloc(size);
    b = (int *) malloc(size);
    c = (int *) malloc(size);

    for (int i=0; i<n*n; ++i) {
        a[i] = b[i] = 1;
        c[i] = 0;
    }
    

    // cudaMalloc(&vecd, size);  // �f�o�C�X�������̊m��
	// cudaMemcpy(vecd, vec, size, cudaMemcpyHostToDevice);
    // cudaEventRecord(start, 0);

    hipMalloc(&ad, size);
    hipMalloc(&bd, size);
    hipMalloc(&cd, size);
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
    hipMemcpy(cd, c, size, hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    

    adm_mult_matrix<<<Dg, Db>>>(ad, bd, cd);

    // cudaMemcpy(vec, vecd, size, cudaMemcpyDeviceToHost);  // ���ʂ̃f�o�C�X����z�X�g�ւ̃R�s�[
    hipMemcpy(a, ad, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, bd, size, hipMemcpyDeviceToHost);
    hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

    // long long sum = 0;
    // for (int i = 0; i < n*n; ++i) {
    //     sum += vec[i];  // ���ʂ̌v�Z
    // }
    // printf("sum = %lld\n", sum);
    printf("c[%d][%d] = %d\n", 0, 0, c[0]);
    printf("c[%d][%d] = %d\n", n-1, n-1, c[n*n-1]);
    

    

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);
    printf("\nCalculation End\n");
    printf("\nProcessing Time : %.3f [msec]\n", timer);

    hipEventDestroy(start);
    hipEventDestroy(end);

    // free(vec);
    // cudaFree(vecd);

    free(a);
    free(b);
    free(c);
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    

    return 0;
}
