#include "hip/hip_runtime.h"
/*
 * main.cu(convert_color.cu) : �F�ϊ�
 * @ KLO lab. in TMU (2013/12/20[Fri])
 */
 

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define HTH hipMemcpyHostToHost
#define HTD hipMemcpyHostToDevice
#define DTH hipMemcpyDeviceToHost
#define DTD hipMemcpyDeviceToDevice

#define Nx 1024
#define Ny 1024

#define Db_x 256
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);

float *h_IMG_input;	     /*  ���̓f�[�^�p(Host)  */
float *h_IMG_output;     /*  �o�̓f�[�^�p(Host)  */

float *d_IMG_input;	     /*  ���̓f�[�^�p(Device)  */
float *d_IMG_output;     /*  �o�̓f�[�^�p(Device)  */

hipEvent_t start, end;	     /*  ���Ԍv���p  */
float timer;	     /*  ���Ԍv���p  */

int element = sizeof(float) * Nx * Ny;

float alpha = 2.f;


/*
 * convert_color.cu : �F�ϊ�
 */

__global__ void convert_upsideDown
(
	float *d_IMG_input,
	float *d_IMG_output
)
{
	int X, Y, ID, toID;

	X = threadIdx.x + (blockIdx.x * blockDim.x);
	Y = threadIdx.y + (blockIdx.y * blockDim.y);
	ID = X + Y * Nx;
	toID = Nx-1-X + Y * Nx;
	if (toID <0 || toID >= Nx*Nx) printf("toID %d\n", toID);
	d_IMG_output[toID] = d_IMG_input[ID];
	// toID = Y+ (Nx-X-1)*Nx;
	
	// d_IMG_output[ID] = d_IMG_input[toID];
}

/*
 * ��������main : �F�ϊ�
 */

int main
(
	int    argc, // Argument Count
	char **argv  // Argument Vector
)
{
	// ��u�҂��Ƃ�GPU������U���Ă���̂ŁC�l��ύX���ĉ�����(0�`3)
	int GPU_Num = 0;

	hipSetDevice(GPU_Num);

	printf("convert_upsideDwon.cu\n");
	printf("Nx = %d, Ny = %d\n", Nx, Ny);
	printf("Number %d GPU working\n", GPU_Num);

	int i, j, ID;

	// �f�o�C�X�iVRAM���j�ɗv�f���m��
	hipMalloc((void **)&d_IMG_input, element);
	hipMalloc((void **)&d_IMG_output, element);

	hipMemset(d_IMG_input, 0, element);
	hipMemset(d_IMG_output, 0, element);

	// �z�X�g�iRAM���j�ɗv�f���m��
	hipHostAlloc((void **)&h_IMG_input, element, hipHostMallocPortable);
	hipHostAlloc((void **)&h_IMG_output, element, hipHostMallocPortable);

	memset(h_IMG_input, 0, element);
	memset(h_IMG_output, 0, element);

//�t�@�C�����f�[�^����
	FILE *fp;	     /*  ���o�t�@�C���p  */

	fp = fopen("./man1024.img", "r");     /*  �Ǎ��݃��[�h�Ńt�@�C�����I�[�v������  */
	if(fp == NULL) {
		printf("�t�@�C�����J�����Ƃ��o���܂���ł����D\n");
		return 0;
	}
 
	for(j = 0; j < Ny; j++){
		for(i = 0; i < Nx; i++){
			ID = i + j * Nx;
			fscanf(fp, "%f", &(h_IMG_input[ID]) );     /*  1�s�ǂށ@�� h_IMG_input[ID])�ɓ���� */
		}
	}
	
	fclose(fp);     /*  �t�@�C�����N���[�Y����  */

	
	// ���Z���Ԃ��v�����邽�߂�hipEventCreate�����s
	hipEventCreate(&start);
	hipEventCreate(&end);

	printf("\nCalculation Start\n");

	hipMemcpy(d_IMG_input, h_IMG_input, element, HTD);

	// ���Z���Ԃ��v�����邽�߂�hipEventRecord�����s���v�Z�̊J�n���L�^
	hipEventRecord(start, 0);


	/* image processing */
	convert_upsideDown <<< Dg, Db >>> (d_IMG_input, d_IMG_output);

	hipDeviceSynchronize();

    // ���Z���Ԃ��v�����邽�߂�hipEventRecord�����s���v�Z�̏I�����L�^
    hipEventRecord(end, 0);

    // ���Z���Ԃ��Z�o
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);

    printf("\nCalculation End\n");

    printf("\nProcessing Time : %.3f [msec]\n", timer);

    /* File Output */
    hipMemcpy(h_IMG_output, d_IMG_output, element, DTH);

//�o�͗p�f�[�^���t�@�C���ɏ�������
    fp = fopen("output1024convert_gpu_5b.img", "w");     /*  �����݃��[�h�Ńt�@�C�����I�[�v������  */
   
    if(fp == NULL){
        printf("�t�@�C�������܂���ł���");
        return 0;  /*  �����Ńv���O�����I��  */
    }

    for(j = 0; j < Ny; j++){
		for(i = 0; i < Nx; i++){
			ID = i + j * Nx;
			fprintf(fp, "%d\n", (unsigned char) h_IMG_output[ID]);     /*  1�s�����݁@�� h_IMG_output[ID])�ɓ���� */
		}
	}

	fclose(fp);

    hipFree(d_IMG_input);
    hipFree(d_IMG_output);

    hipHostFree(h_IMG_input);
    hipHostFree(h_IMG_output);

    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
