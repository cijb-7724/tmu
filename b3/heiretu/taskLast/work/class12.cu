
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloThread() {
    printf("hello thread. %d %d\n", blockIdx.x, threadIdx.x);
    printf("%d \n", blockIdx.x * blockDim.x + threadIdx.x);
}
int main() {
    helloThread<<<2, 4>>>();
    hipDeviceSynchronize();
    return 0;
}