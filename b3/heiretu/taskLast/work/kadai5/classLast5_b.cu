#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define HTH hipMemcpyHostToHost
#define HTD hipMemcpyHostToDevice
#define DTH hipMemcpyDeviceToHost
#define DTD hipMemcpyDeviceToDevice

#define Nx 1024
#define Ny 1024

#define Db_x 256
#define Db_y 1
#define Db_z 1

#define Dg_x (Nx / Db_x)
#define Dg_y (Ny / Db_y)
#define Dg_z 1

dim3 Db(Db_x, Db_y, Db_z);
dim3 Dg(Dg_x, Dg_y, Dg_z);

float *h_IMG_input;    /*  入力画像のポインタ(Host)  */
float *h_IMG_output;   /*  出力画像のポインタ(Host)  */

float *d_IMG_input;    /*  入力画像のポインタ(Device)  */
float *d_IMG_output;   /*  出力画像のポインタ(Device)  */

hipEvent_t start, end;  /*  測定用イベント  */
float timer;             /*  時間計測結果  */

int element = sizeof(float) * Nx * Ny;

/*
 * convert_upsideDown.cu : 上下反転するCUDAカーネル
 */

__global__ void convert_upsideDown(float *d_IMG_input, float *d_IMG_output) {
    int X, Y, ID, toID;

    X = threadIdx.x + (blockIdx.x * blockDim.x);
    Y = threadIdx.y + (blockIdx.y * blockDim.y);
    ID = X + Y * Nx;
    toID = Nx - 1 - X + Y * Nx;
    d_IMG_output[toID] = d_IMG_input[ID];
}

int main(int argc, char **argv) {
    // 使用するGPUデバイス番号
    int GPU_Num = 0;
    hipSetDevice(GPU_Num);

    printf("convert_upsideDown.cu\n");
    printf("Nx = %d, Ny = %d\n", Nx, Ny);
    printf("Number %d GPU working\n", GPU_Num);

    // ホストとデバイスのメモリ確保
    hipMalloc((void **)&d_IMG_input, element);
    hipMalloc((void **)&d_IMG_output, element);
    hipHostAlloc((void **)&h_IMG_input, element, hipHostMallocPortable);
    hipHostAlloc((void **)&h_IMG_output, element, hipHostMallocPortable);

    memset(h_IMG_input, 0, element);
    memset(h_IMG_output, 0, element);

    // ファイル読み込み
    FILE *fp;
    fp = fopen("./man1024.img", "r");
    if (fp == NULL) {
        printf("ファイルを開けませんでした\n");
        return 0;
    }
    for (int j = 0; j < Ny; j++) {
        for (int i = 0; i < Nx; i++) {
            int ID = i + j * Nx;
            fscanf(fp, "%f", &(h_IMG_input[ID]));
        }
    }
    fclose(fp);

    // 計算時間測定開始
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    // デバイスへデータ転送およびカーネル実行
    hipMemcpy(d_IMG_input, h_IMG_input, element, HTD);
    convert_upsideDown<<<Dg, Db>>>(d_IMG_input, d_IMG_output);
    hipDeviceSynchronize();

    // 計算時間測定終了
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&timer, start, end);
    printf("\nCalculation End\n");
    printf("\nProcessing Time : %.3f [msec]\n", timer);

    // 結果をホストにコピーしてファイルに書き出し
    hipMemcpy(h_IMG_output, d_IMG_output, element, DTH);
    fp = fopen("output1024convert_gpu_5b.img", "w");
    if (fp == NULL) {
        printf("ファイルを開けませんでした\n");
        return 0;
    }
    for (int j = 0; j < Ny; j++) {
        for (int i = 0; i < Nx; i++) {
            int ID = i + j * Nx;
            fprintf(fp, "%d\n", (unsigned char)h_IMG_output[ID]);
        }
    }
    fclose(fp);

    // メモリ解放
    hipFree(d_IMG_input);
    hipFree(d_IMG_output);
    hipHostFree(h_IMG_input);
    hipHostFree(h_IMG_output);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
