#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>


#define GRID_SIZE 128    // グリッドのサイズ
#define BLOCK_SIZE 64    // ブロックのサイズ

#define Nx (BLOCK_SIZE * GRID_SIZE)  // X方向の要素数
#define Ny (BLOCK_SIZE * GRID_SIZE)  // Y方向の要素数

#define Db_x BLOCK_SIZE  // X方向のブロックサイズ
#define Db_y 1            // Y方向のブロックサイズ
#define Db_z 1            // Z方向のブロックサイズ

#define Dg_x (Nx / Db_x)  // X方向のグリッド数
#define Dg_y (Ny / Db_y)  // Y方向のグリッド数
#define Dg_z 1            // Z方向のグリッド数

dim3 Db(Db_x, Db_y, Db_z);  // 3次元ブロック構造
dim3 Dg(Dg_x, Dg_y, Dg_z);  // 3次元グリッド構造

hipEvent_t start, end;  // CUDAイベント変数
float timer;             // 処理時間を格納する変数

// カーネル関数：行列の要素ごとの計算を実行
__global__ void adder(long *vecd) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // 行番号の計算
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // 列番号の計算
    vecd[i * Nx + j] = (i - j) * (i - j);           // 要素ごとの計算
}

int main(int argc, char **argv) {
    hipSetDevice(0);  // CUDAデバイスの設定

    long *vec, *vecd;  // ホストとデバイスのメモリ領域を指すポインタ
    int n = GRID_SIZE * BLOCK_SIZE;  // 行列の1辺のサイズ
    int size = n * n * sizeof(long); // 行列の総要素数

    hipEventCreate(&start);  // 開始時間を記録するCUDAイベントの生成
    hipEventCreate(&end);    // 終了時間を記録するCUDAイベントの生成

    printf("\nCalculation Start\n");

    vec = (long *)malloc(size);  // ホスト側のメモリ領域の確保

    hipMalloc(&vecd, size);  // デバイス側のメモリ領域の確保
    hipMemcpy(vecd, vec, size, hipMemcpyHostToDevice);  // ホストからデバイスへのデータ転送
    hipEventRecord(start, 0);  // 計算開始時間の記録

    adder<<<Dg, Db>>>(vecd);  // カーネル関数の実行

    hipMemcpy(vec, vecd, size, hipMemcpyDeviceToHost);  // デバイスからホストへのデータ転送

    long long sum = 0;  // 結果の合計値
    for (int i = 0; i < n * n; ++i) {
        sum += vec[i];  // 合計の計算
    }
    printf("L   = %d\n", GRID_SIZE * BLOCK_SIZE);  // サイズの表示
    printf("sum = %lld\n", sum);                   // 合計の表示

    hipEventRecord(end, 0);            // 計算終了時間の記録
    hipEventSynchronize(end);          // イベントの完了を待機
    hipEventElapsedTime(&timer, start, end);  // 計算時間の算出
    printf("\nCalculation End\n");
    printf("\nProcessing Time : %.3f [msec]\n", timer);  // 計算時間の表示

    hipEventDestroy(start);  // 開始イベントの破棄
    hipEventDestroy(end);    // 終了イベントの破棄

    free(vec);         // ホスト側メモリの解放
    hipFree(vecd);
}
