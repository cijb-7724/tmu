#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


// 定数定義
#define GRID_SIZE 64
#define BLOCK_SIZE 32

#define MATRIX_SIZE (BLOCK_SIZE * GRID_SIZE)

// グリッドとブロックの次元設定
dim3 Db(BLOCK_SIZE, 1, 1); // ブロックの次元
dim3 Dg(GRID_SIZE, GRID_SIZE, 1); // グリッドの次元

// CUDAイベント変数の宣言
hipEvent_t start, end;
float timer;

// カーネル関数: 2つの行列の乗算
__global__ void mult_matrix(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // 行インデックス
    int j = blockIdx.y * blockDim.y + threadIdx.y; // 列インデックス
    for (int k = 0; k < MATRIX_SIZE; ++k) {
        // 行列の各要素の計算
        c[i * MATRIX_SIZE + j] += a[i * MATRIX_SIZE + k] * b[k * MATRIX_SIZE + j];
    }
}

int main() {
    hipSetDevice(0); // CUDAデバイスの設定

    int *a, *b, *c; // ホスト側の行列
    int *ad, *bd, *cd; // デバイス側の行列
    int size = MATRIX_SIZE * MATRIX_SIZE * sizeof(int); // 行列のサイズ

    hipEventCreate(&start); // 開始時刻のCUDAイベントの作成
    hipEventCreate(&end); // 終了時刻のCUDAイベントの作成

    printf("Matrix Multiplication\n");
    printf("\nCalculation Start\n");

    // ホスト側の行列メモリの割り当てと初期化
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for (int i = 0; i < MATRIX_SIZE * MATRIX_SIZE; ++i) {
        a[i] = b[i] = 1; // すべての要素を1に初期化
        c[i] = 0; // cの要素を0に初期化
    }

    // デバイス側の行列メモリの割り当てと初期化
    hipMalloc(&ad, size);
    hipMalloc(&bd, size);
    hipMalloc(&cd, size);
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
    hipMemcpy(cd, c, size, hipMemcpyHostToDevice);

    hipEventRecord(start, 0); // 演算開始時刻の記録

    mult_matrix<<<Dg, Db>>>(ad, bd, cd); // カーネル関数の呼び出し

    hipMemcpy(c, cd, size, hipMemcpyDeviceToHost); // 結果のデバイス側からホスト側への転送

    // 結果の表示
    printf("c[%d][%d] = %d\n", 0, 0, c[0]);
    printf("c[%d][%d] = %d\n", MATRIX_SIZE - 1, MATRIX_SIZE - 1, c[MATRIX_SIZE * MATRIX_SIZE - 1]);

    hipEventRecord(end, 0); // 演算終了時刻の記録
    hipEventSynchronize(end); // イベントの同期
    hipEventElapsedTime(&timer, start, end); // 演算時間の計算
    printf("\nCalculation End\n");
    printf("\nProcessing Time : %.3f [msec]\n", timer); // 演算時間の表示

    hipEventDestroy(start); // 開始時刻のCUDAイベントの破棄
    hipEventDestroy(end); // 終了時刻のCUDAイベントの破棄

    // メモリの解放
    free(a);
    free(b);
    free(c);
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);

    return 0;
}
