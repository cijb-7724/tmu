#include "hip/hip_runtime.h"
#include <stdio.h>

#define grid 1e4
#define block 1e3

__global__ void vecAdd(double *Ad, double *Bd, double *Cd) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    Cd[i] = Ad[i] + Bd[i];
}


int main() {
    hipSetDevice(0);
    double *A, *Ad, *B, *Bd, *C, *Cd;;
    int i, N = Grid*block, size = grid*block*sizeof(double):
}